#include "hip/hip_runtime.h"
/* Copyright 2011 Russel Steinbach, Jeffrey Blanchard, Bradley Gordon,
 *   and Toluwaloju Alabi
 *   Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <thrust/pair.h>
#include <thrust/extrema.h>
#include <thrust/partition.h>
#include <iostream>
using namespace std;

#define MAX_THREADS_PER_BLOCK 1024
#define CUTOFF_POINT 2200000 

#define CUDA_CALL(x) do { if((x) != hipSuccess) {      \
      printf("Error at %s:%d\n",__FILE__,__LINE__);     \
      return EXIT_FAILURE;}} while(0)

  template<typename T>
  void cleanup(uint *h_c, T* d_k, int *etb, uint *bc){
    free(h_c);
    hipFree(d_k);
    hipFree(etb);
    hipFree(bc);
  }

//This function initializes a vector to all zeros on the host (CPU)
void setToAllZero(uint* deviceVector, int length){
  hipMemset(deviceVector, 0, length * sizeof(uint));
 }

//this function assigns elements to buckets
template <typename T>
__global__ void assignBucket(T* d_vector, int length, int bucketNumbers, double slope, double minimum, int* bucket, uint* bucketCount, int offset){
  
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int bucketIndex;
  extern __shared__ uint sharedBuckets[];
  int index = threadIdx.x;  
 
  //variables in shared memory for fast access
  __shared__ int sbucketNums;
  __shared__ double sMin;
  sbucketNums = bucketNumbers;
  sMin = minimum;

  //reading bucket counts into shared memory where increments will be performed
  if(index < bucketNumbers){
    sharedBuckets[index] = 0;
  }
  __syncthreads();

  //assigning elements to buckets and incrementing the bucket counts
  if(idx < length)    {
      int i;
      for(i=idx; i< length; i+=offset){   
          //calculate the bucketIndex for each element
          bucketIndex =  (d_vector[i] - sMin) * slope;

          //if it goes beyond the number of buckets, put it in the last bucket
          if(bucketIndex >= sbucketNums){
            bucketIndex = sbucketNums - 1;
          }
          bucket[i] = bucketIndex;
          atomicInc(&sharedBuckets[bucketIndex], length);
        }
    }

  __syncthreads();

  //reading bucket counts from shared memory back to global memory
  if(index < bucketNumbers){
    atomicAdd(&bucketCount[index], sharedBuckets[index]);
  }
}

//this function reassigns elements to buckets
template <typename T>
__global__ void reassignBucket(T* d_vector, int *bucket, uint *bucketCount, const int bucketNumbers, const int length, const double slope, const double maximum, const double minimum, int offset, int Kbucket){
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  extern __shared__ uint sharedBuckets[];
  int index = threadIdx.x;
  int bucketIndex;

  //reading bucket counts to shared memory where increments will be performed
  if(index < bucketNumbers){
    sharedBuckets[index] =0;
  }
  __syncthreads();

  //assigning elements to buckets and incrementing the bucket counts
  if (idx < length){
    int i;

    for(i=idx; i<length; i+=offset){
      if(bucket[i] != Kbucket){
        bucket[i] = bucketNumbers+1;
      }
      else{
        //calculate the bucketIndex for each element
        bucketIndex = (d_vector[i] - minimum) * slope;

        //if it goes beyond the number of buckets, put it in the last bucket
        if(bucketIndex >= bucketNumbers){
          bucketIndex = bucketNumbers - 1;
        }
        bucket[i] = bucketIndex;

        atomicInc(&sharedBuckets[bucketIndex], length);
      }
    }
  }

  __syncthreads();

  //reading bucket counts from shared memory back to global memory
  if(index < bucketNumbers){
    atomicAdd(&bucketCount[index], sharedBuckets[index]);
  }
}

//copy elements in the kth bucket to a new array
template <typename T>
__global__ void copyElement(T* d_vector, int length, int* elementArray, int bucket, T* newArray, uint* count, int offset){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < length){
    int i;
    for(i=idx; i<length; i+=offset){
      //copy elements in the kth bucket to the new array
      if(elementArray[i] == bucket){
        newArray[atomicInc(&count[0],length)] = d_vector[i];
      }
    }
  }
}

//this function finds the bin containing the kth element we are looking for (works on the host)
  inline int FindKBucket(uint *d_counter, uint *h_counter, const int num_buckets, const int k, uint * sum){
    hipMemcpy(sum, d_counter, sizeof(uint), hipMemcpyDeviceToHost);
    int Kbucket = 0;
    
    if (sum[0]<k){
      hipMemcpy(h_counter, d_counter, num_buckets * sizeof(uint), hipMemcpyDeviceToHost);
      while ( (sum[0]<k) & (Kbucket<num_buckets-1)){
        Kbucket++; 
        sum[0] = sum[0] + h_counter[Kbucket];
      }
    }
    else{
      hipMemcpy(h_counter, d_counter, sizeof(uint), hipMemcpyDeviceToHost);
    }
  
    return Kbucket;
  }

template <typename T>
__global__ void GetKvalue(T* d_vector, int * d_bucket, const int Kbucket, const int n, T* Kvalue, int offset )
{
  uint xIndex = blockDim.x * blockIdx.x + threadIdx.x;

  if (xIndex < n) {
    int i;
    for(i=xIndex; i<n; i+=offset){
      if ( d_bucket[i] == Kbucket ) {
        Kvalue[0] = d_vector[i];
      }
    }
  }
}


/************************************************************************/
/************************************************************************/
//THIS IS THE PHASE TWO FUNCTION WHICH WILL BE CALLED IF THE INPUT
//LENGTH IS LESS THAN THE CUTOFF OF 2MILLION 200 THOUSAND
/************************************************************************/


template <typename T>
T phaseTwo(T* d_vector, int length, int K, int blocks, int threads, double maxValue = 0, double minValue = 0){ 
  //declaring and initializing variables for kernel launches
  int threadsPerBlock = threads;
  int numBlocks = blocks;
  int numBuckets = 1024;
  int offset = blocks * threads;

  uint sum=0, Kbucket=0, iter=0;
  int Kbucket_count = 0;
 
  //initializing variables for kernel launches
  if(length < 1024){
    numBlocks = 1;
  }
  //variable to store the end result
  T kthValue =0;

  //declaring and initializing other variables
  size_t size = length * sizeof(int);
  size_t totalBucketSize = numBuckets * sizeof(uint);

  //allocate memory to store bucket assignments and to count elements in buckets
  int* elementToBucket;
  uint* d_bucketCount;
  hipMalloc(&elementToBucket, size);
  hipMalloc(&d_bucketCount, totalBucketSize);
  uint * h_bucketCount = (uint*)malloc(totalBucketSize);

  T* d_Kth_val;
  hipMalloc(&d_Kth_val, sizeof(T));

  thrust::device_ptr<T>dev_ptr(d_vector);
  //if max == min, then we know that it must not have had the values passed in. 
  if(maxValue == minValue){
    thrust::pair<thrust::device_ptr<T>, thrust::device_ptr<T> > result = thrust::minmax_element(dev_ptr, dev_ptr + length);
    minValue = *result.first;
    maxValue = *result.second;
  }
  double slope = (numBuckets - 1)/(maxValue - minValue);
  //first check is max is equal to min
  if(maxValue == minValue){
    cleanup(h_bucketCount, d_Kth_val, elementToBucket,d_bucketCount);
    return maxValue;
  }

  //make all entries of this vector equal to zero
  setToAllZero(d_bucketCount, numBuckets);
  //distribute elements to bucket
  assignBucket<<<numBlocks, threadsPerBlock, numBuckets*sizeof(uint)>>>(d_vector, length, numBuckets, slope, minValue, elementToBucket, d_bucketCount, offset);

  //find the bucket containing the kth element we want
  Kbucket = FindKBucket(d_bucketCount, h_bucketCount, numBuckets, K, &sum);
  Kbucket_count = h_bucketCount[Kbucket];

  while ( (Kbucket_count > 1) && (iter < 1000)){
    minValue = max(minValue, minValue + Kbucket/slope);
    maxValue = min(maxValue, minValue + 1/slope);

    K = K - sum + Kbucket_count;

    if ( maxValue - minValue > 0.0f ){
      slope = (numBuckets - 1)/(maxValue-minValue);
      setToAllZero(d_bucketCount, numBuckets);
      reassignBucket<<< numBlocks, threadsPerBlock, numBuckets * sizeof(uint) >>>(d_vector, elementToBucket, d_bucketCount, numBuckets,length, slope, maxValue, minValue, offset, Kbucket);

      sum = 0;
      Kbucket = FindKBucket(d_bucketCount, h_bucketCount, numBuckets, K, &sum);
      Kbucket_count = h_bucketCount[Kbucket];

      iter++;
    }
    else{
      //if the max and min are the same, then we are done
      cleanup(h_bucketCount, d_Kth_val, elementToBucket, d_bucketCount);
      return maxValue;
    }
  }

    GetKvalue<<<numBlocks, threadsPerBlock >>>(d_vector, elementToBucket, Kbucket, length, d_Kth_val, offset);
    hipMemcpy(&kthValue, d_Kth_val, sizeof(T), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  

  cleanup(h_bucketCount, d_Kth_val, elementToBucket, d_bucketCount);
  return kthValue;
}



/* this function finds the kth-largest element from the input array */
template <typename T>
T phaseOne(T* d_vector, int length, int K, int blocks, int threads, int pass = 0){
  //declaring variables for kernel launches
  int threadsPerBlock = threads;
  int numBlocks = blocks;
  int numBuckets = 1024;
  int offset = blocks * threads;
  int kthBucket, kthBucketCount;
  int newInputLength;
  int* elementToBucket; //array showing what bucket every element is in
  //declaring and initializing other variables

  uint *d_bucketCount, *count; //array showing the number of elements in each bucket
  uint kthBucketScanSize = 0;

  size_t size = length * sizeof(int);

  //variable to store the end result
  T kthValue = 0;
  T* newInput;

  //find max and min with thrust
  double maximum, minimum;

  thrust::device_ptr<T>dev_ptr(d_vector);
  thrust::pair<thrust::device_ptr<T>, thrust::device_ptr<T> > result = thrust::minmax_element(dev_ptr, dev_ptr + length);

  minimum = *result.first;
  maximum = *result.second;

  //if the max and the min are the same, then we are done
  if(maximum == minimum){
    return maximum;
  }
  //if we want the max or min just return it
  if(K == 1){
    return minimum;
  }
  if(K == length){
    return maximum;
  }		
  //Allocate memory to store bucket assignments
  
  CUDA_CALL(hipMalloc(&elementToBucket, size));

  //Allocate memory to store bucket counts
  size_t totalBucketSize = numBuckets * sizeof(uint);
  CUDA_CALL(hipMalloc(&d_bucketCount, totalBucketSize));
  uint* h_bucketCount = (uint*)malloc(totalBucketSize);

  //Calculate max-min
  double range = maximum - minimum;
  //Calculate the slope, i.e numBuckets/range
  double slope = (numBuckets - 1)/range;

  hipMalloc(&count, sizeof(uint));
  //Set the bucket count vector to all zeros
  setToAllZero(d_bucketCount, numBuckets);

  //Distribute elements into their respective buckets
  assignBucket<<<numBlocks, threadsPerBlock, numBuckets*sizeof(uint)>>>(d_vector, length, numBuckets, slope, minimum, elementToBucket, d_bucketCount, offset);
  kthBucket = FindKBucket(d_bucketCount, h_bucketCount, numBuckets, K, & kthBucketScanSize);
  kthBucketCount = h_bucketCount[kthBucket];
 

  //we must update K since we have reduced the problem size to elements in the kth bucket
  if(kthBucket != 0){
    K = kthBucketCount - (kthBucketScanSize - K);
  }

  //copy elements in the kth bucket to a new array
  hipMalloc(&newInput, kthBucketCount * sizeof(T));
  setToAllZero(count, 1);
  copyElement<<<numBlocks, threadsPerBlock>>>(d_vector, length, elementToBucket, kthBucket, newInput, count, offset);


  //store the length of the newly copied elements
  newInputLength = kthBucketCount;


  //if we only copied one element, then we are done
  if(newInputLength == 1){
    thrust::device_ptr<T>new_ptr(newInput);
    kthValue = new_ptr[0];
      
    //free all used memory
    hipFree(elementToBucket); hipFree(d_bucketCount); hipFree(count); hipFree(newInput);
    return kthValue;
  }
 
  /*********************************************************************/
  //END OF FIRST PASS, NOW WE PROCEED TO SUBSEQUENT PASSES
  /*********************************************************************/

  //if the new length is greater than the CUTOFF, run the regular phaseOne again
  if(newInputLength > CUTOFF_POINT && pass < 1){
    if(pass > 0){
      hipFree(d_vector);
    }
    hipFree(elementToBucket);  hipFree(d_bucketCount); hipFree(count);
    kthValue = phaseOne(newInput, newInputLength, K, blocks, threads,pass + 1);
  }
  else{
    minimum = max(minimum, minimum + kthBucket/slope);
    maximum = min(maximum, minimum + 1/slope);
    kthValue = phaseTwo(newInput,newInputLength, K, blocks, threads,maximum, minimum);
  }


  //free all used memory
  hipFree(elementToBucket);  hipFree(d_bucketCount); hipFree(newInput); hipFree(count);

  return kthValue;
}

/**************************************************************************/
/**************************************************************************/
//THIS IS THE BUCKETSELECT FUNCTION WRAPPER THAT CHOOSES THE CORRECT VERSION
//OF BUCKET SELECT TO RUN BASED ON THE INPUT LENGTH
/**************************************************************************/
template <typename T>
T bucketSelectWrapper(T* d_vector, int length, int K, int blocks, int threads)
{
  T kthValue;
  //change K to be the kth smallest
  K = length - K + 1;

  if(length <= CUTOFF_POINT)
    {
      kthValue = phaseTwo(d_vector, length, K, blocks, threads);
      return kthValue;
    }
  else
    {
      kthValue = phaseOne(d_vector, length, K, blocks, threads);
      return kthValue;
    }

}



extern "C" {
struct le
  {
    double pivot;

    le(double pivot)
    {
      this->pivot = pivot;
    }

    __host__ __device__
    bool operator()(const double &x)
    {
      return x <= pivot;
    }
  };

  struct le_by_key
  {
    double pivot;

    le_by_key(double pivot)
    {
      this->pivot = pivot;
    }

    __host__ __device__
    bool operator()(const thrust::tuple<int64_t, double> &x)
    {
      return thrust::get<1>(x) <= pivot;
    }
  };

  void partition_double(double *input, int64_t length, double pivot)
  {
    thrust::device_ptr<double> data(input);
    le predicate(pivot);
    thrust::partition(data, data+length, predicate);
  }

  int64_t partition_int_by_key(int64_t *values, double *keys, int64_t length, double pivot)
  {
    le_by_key predicate(pivot);
    thrust::device_ptr<int64_t> values_d(values);
    thrust::device_ptr<double> keys_d(keys);
    thrust::zip_iterator<thrust::tuple<thrust::device_ptr<int64_t>, thrust::device_ptr<double> > > start = thrust::make_zip_iterator(thrust::make_tuple(values_d, keys_d));
    return thrust::partition(start, start+length, predicate) - start;
  }
  
  void top_k_double(double* d_vector, int length, int k, int blocks, int threads) {
    thrust::device_ptr<double> dp = thrust::min_element(thrust::device_ptr<double>(d_vector), thrust::device_ptr<double>(d_vector)+length);
    double min, first;
    thrust::copy(dp, dp+1, &min);
    double pivot =  bucketSelectWrapper(d_vector, length, k, blocks, threads);
    partition_double(d_vector, length, pivot);
  }

  void top_k_int_by_key(int64_t* values, double* keys, int length, int k, int blocks, int threads) {
    thrust::device_ptr<double> dp = thrust::min_element(thrust::device_ptr<double>(keys), thrust::device_ptr<double>(keys)+length);
    double pivot =  bucketSelectWrapper(keys, length, k, blocks, threads);
    partition_int_by_key(values, keys, length, pivot);
  }
}

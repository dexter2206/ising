
#include <hip/hip_runtime.h>
#include <stdio.h>

template <typename T>
__device__
T energy(int64_t state_repr, T* Q, int N)
{
    int i, j;
    char bits[64];
    T energy = 0;

    for(i = 0; i < N; i++) {
        bits[i] = (state_repr >> i) & 1;
    }

    for(i = 0; i < N; i++) {
        if(bits[i]) {
            for(j = i; j < N; j++) {
                energy -= Q[i * N + j] * bits[i] * bits[j];
            }
        }
    }
    return energy;
}


template <typename T>
void __global__ search(T* Q, int N, int sweep_size, T* energies, long int* states, long int m)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int i;
    long int state_repr;
    if(idx < sweep_size) {
        state_repr = idx + m;
        states[idx] = state_repr;
        energies[idx] = energy(state_repr, Q, N);
    }
}


template <typename T>
void __global__ search_energies_only(T* Q, int N, int sweep_size, T* energies, long int m)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int i;
    long int state_repr;
    if(idx < sweep_size) {
        state_repr = idx + m;
        energies[idx] = energy(state_repr, Q, N);
    }
}


template
__global__
void search<float>(float* Q, int N, int sweep_size, float* energies, long int* states, long int m);


template
__global__
void search<double>(double* Q, int N, int sweep_size, double* energies, long int* states, long int m);


template
__global__
void search_energies_only<float>(float* Q, int N, int sweep_size, float* energies, long int m);


template
__global__
void search_energies_only<double>(double* Q, int N, int sweep_size, double* energies, long int m);

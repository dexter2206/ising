
#include <hip/hip_runtime.h>
#include <stdio.h>

template <typename T>
__device__
T energy(int64_t state_repr, T* Q, int N)
{
  int i, j;
  short int bit;
  T energy = 0;
  
  for(i = 0; i < N; i++) {
    bit = (state_repr >> i) & 1;
    if(bit) {
      for(j = i; j < N; j++) {
	energy -= Q[i * N + j] * bit * ((state_repr >> j) & 1);
      }
    }
  }
  return energy;
}


template <typename T>
void __global__ search(T* Q, int N, int sweep_size, T* energies, long int* states, long int m)
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int i;
  long int state_repr;
  if(idx < sweep_size) {
    state_repr = idx + m;
    states[idx] = state_repr;
    energies[idx] = energy(state_repr, Q, N);
  }
}


template <typename T>
void __global__ search_energies_only(T* Q, int N, int sweep_size, T* energies, long int m)
{
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int i;
  long int state_repr;
  if(idx < sweep_size) {
    state_repr = idx + m;
    energies[idx] = energy(state_repr, Q, N);
  }    
}


template
__global__
void search<float>(float* Q, int N, int sweep_size, float* energies, long int* states, long int m);


template
__global__
void search<double>(double* Q, int N, int sweep_size, double* energies, long int* states, long int m);


template
__global__
void search_energies_only<float>(float* Q, int N, int sweep_size, float* energies, long int m);


template
__global__
void search_energies_only<double>(double* Q, int N, int sweep_size, double* energies, long int m);

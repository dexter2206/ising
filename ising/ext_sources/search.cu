#include "hip/hip_runtime.h"
#include <kernels.cu>
#include <search.cuh>
#include <stdio.h>
#include <select.cu>


template <typename T>
void find_lowest(
    T* Q,
    int N,
    int chunk_exp,
    T* en_out,
    long int* st_out,
    int num_st,
    int grid_size,
    int block_size,
    void* user_data,
    callback_function callback)
{
    int chunk_size = pow(2, chunk_exp);

    T* d_energies;
    T* d_Jh;
    long int* d_states;
    long int idx;

    hipMalloc((void **) &d_energies, sizeof(T) * chunk_size);
    hipMalloc((void **) &d_states, sizeof(long int) * chunk_size);
    hipMalloc((void **) &d_Jh, sizeof(T) * N * N);
    hipMemcpy(d_Jh, Q, sizeof(T) * N * N, hipMemcpyHostToDevice);

    if(num_st > chunk_size) {
        num_st = chunk_size;
    }

    T* d_low_en;
    long int* d_low_st;

    hipMalloc((void **) &d_low_en, sizeof(T) * num_st * 2);
    hipMalloc((void **) &d_low_st, sizeof(long int) * num_st *2);

    for(long int m=0; m < pow(2, N - chunk_exp); m++) {
        if(callback != NULL) {
            if(callback(m, user_data) == -1) return;
        }
        idx = m * chunk_size;
        search<<<grid_size, block_size>>>(d_Jh, N, chunk_size, d_energies, d_states, idx);

        top_k_int_by_key(d_states, d_energies, chunk_size, chunk_size-num_st, 40, 1024);

        if(m == 0) {
            hipMemcpy(d_low_en, d_energies, num_st * sizeof(T), hipMemcpyDeviceToDevice);
            hipMemcpy(d_low_st, d_states, num_st * sizeof(long int), hipMemcpyDeviceToDevice);
        } else {
            hipMemcpy(d_low_en + num_st, d_energies, num_st * sizeof(T), hipMemcpyDeviceToDevice);
            hipMemcpy(d_low_st + num_st, d_states, num_st * sizeof(long int), hipMemcpyDeviceToDevice);
            top_k_int_by_key(d_low_st, d_low_en, 2 * num_st, num_st, 40, 1024);
        }
    }
    hipMemcpy(en_out, d_low_en, num_st * sizeof(T), hipMemcpyDeviceToHost);
    hipMemcpy(st_out, d_low_st, num_st * sizeof(long int), hipMemcpyDeviceToHost);

    hipFree(d_Jh);
    hipFree(d_energies);
    hipFree(d_states);
    hipFree(d_low_en);
    hipFree(d_low_st);
}


template <typename T>
void find_lowest_energies_only(
    T* Q,
    int N,
    int chunk_exp,
    T* en_out,
    int num_st,
    int grid_size,
    int block_size,
    void * user_data,
    callback_function callback	       )
{
    int chunk_size = pow(2, chunk_exp);

    T* d_energies;
    T* d_Jh;
    long int idx;

    hipMalloc((void **) &d_energies, sizeof(T) * chunk_size);
    hipMalloc((void **) &d_Jh, sizeof(T) * N * N);
    hipMemcpy(d_Jh, Q, sizeof(T) * N * N, hipMemcpyHostToDevice);

    if(num_st > chunk_size) {
        num_st = chunk_size;
    }

    T* d_low_en;
    long int* d_low_st;

    hipMalloc((void **) &d_low_en, sizeof(T) * num_st * 2);

    for(int m=0; m < pow(2, N - chunk_exp); m++) {
        if(callback != NULL) {
            if(callback(m, user_data) == -1) return;
        }
        idx = m * chunk_size;
        search_energies_only<<<grid_size, block_size>>>(d_Jh, N, chunk_size, d_energies, idx);

        top_k(d_energies, chunk_size, chunk_size-num_st, 40, 1024);

        if(m == 0) {
            hipMemcpy(d_low_en, d_energies, num_st * sizeof(T), hipMemcpyDeviceToDevice);
        } else {
            hipMemcpy(d_low_en + num_st, d_energies, num_st * sizeof(T), hipMemcpyDeviceToDevice);
            top_k(d_low_en, 2 * num_st, num_st, 40, 1024);
        }
    }
    hipMemcpy(en_out, d_low_en, num_st * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(d_Jh);
    hipFree(d_energies);
    hipFree(d_low_en);
}


template
void find_lowest<float>(
    float* Jh,
    int N,
    int chunk_exp,
    float* en_out,
    long int* st_out,
    int num_st,
    int grid_size,
    int block_size,
    void* user_data,
    callback_function callback
);

template
void find_lowest<double>(
    double* Jh,
    int N,
    int chunk_exp,
    double* en_out,
    long int* st_out,
    int num_st,
    int grid_size,
    int block_size,
    void* user_data,
    callback_function callback
);


template
void find_lowest_energies_only<float>(
    float* Jh,
    int N,
    int chunk_exp,
    float* en_out,
    int num_st,
    int grid_size,
    int block_size,
    void* user_data,
    callback_function callback
);

template
void find_lowest_energies_only<double>(
    double* Jh,
    int N,
    int chunk_exp,
    double* en_out,
    int num_st,
    int grid_size,
    int block_size,
    void* user_data,
    callback_function callback
);

void getGPUMemInfo(unsigned long* free, unsigned long* total)
{
    hipMemGetInfo(free, total);
}
